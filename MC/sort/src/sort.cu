#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

#include "sort.h"

#define MAX_STR_LEN 30
#define MIN_CHAR 65
#define MAX_CHAR 122



bool check_cuda_error(hipError_t status) {
    if (status != hipSuccess) {
        cout << "CUDA Error: " << hipGetErrorString(status) << endl;
        return false;
    }
    return true;
}

void print_mem(){
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    printf("GPU memory: free = %zu bytes, total = %zu bytes\n", free_mem, total_mem);
}


__device__ int __strncmp_kernel(const char *str_1, const char *str_2, size_t n) {
    while (n--) {
        if (*str_1 != *str_2) {
            return *(unsigned char *)str_1 - *(unsigned char *)str_2;
        }
        if (*str_1 == '\0') {
            break;
        }
        str_1++;
        str_2++;
    }
    return 0;
}

__device__ int __strlen_kernel(const char *str) {
    int len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

__device__ int __char_to_index_kernel(char ch){
    return (int)ch - MIN_CHAR+1;
}

__device__ char __index_to_char_kernel(int n){
    return (char)(n + MIN_CHAR-1);
}



__global__ void __check_sorted_arr_kernel(int *N, char **str_arr, char **sorted_arr, int *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < *N) {
        if (__strncmp_kernel(*(str_arr+idx), *(sorted_arr+idx), MAX_STR_LEN) != 0) {
            atomicAdd(result, 1);
        }
    }
}

__global__ void __test(char *str) {
    printf("%c\n", *(str+1));
}

int gpu_check_sorted_arr(int block_size, int N, char **str_arr, char **sorted_arr) {
    if (N <= 0 || block_size <= 0 || str_arr == NULL || sorted_arr == NULL) 
        return -1;

    char **d_str_arr, **d_sorted_arr, *d_str;
    int *d_result, *d_N;
    int result;

    check_cuda_error(hipMalloc((void**)&d_str_arr, N * sizeof(char *)));
    check_cuda_error(hipMalloc((void**)&d_sorted_arr, N * sizeof(char *)));
    check_cuda_error(hipMalloc((void**)&d_result, sizeof(int)));
    check_cuda_error(hipMalloc((void**)&d_N, sizeof(int)));
    for(int i=0; i<N; i++) {
        check_cuda_error(hipMalloc((void**)&d_str, sizeof(char) * MAX_STR_LEN));
        check_cuda_error(hipMemcpy(d_str, str_arr[i], sizeof(char) * MAX_STR_LEN, hipMemcpyHostToDevice));
        check_cuda_error(hipMemcpy(d_str_arr+i, &d_str, sizeof(char*), hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void**)&d_str, sizeof(char) * MAX_STR_LEN));
        check_cuda_error(hipMemcpy(d_str, sorted_arr[i], sizeof(char) * MAX_STR_LEN, hipMemcpyHostToDevice));
        check_cuda_error(hipMemcpy(d_sorted_arr+i, &d_str, sizeof(char*), hipMemcpyHostToDevice));
    }
    check_cuda_error(hipMemset(d_result, 0, sizeof(int)));
    check_cuda_error(hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice));

    int block_num = (N + block_size - 1) / block_size;
    __check_sorted_arr_kernel<<<block_num, block_size>>>(d_N, d_str_arr, d_sorted_arr, d_result);
    check_cuda_error(hipDeviceSynchronize());

    check_cuda_error(hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost));

    // for (int i=0; i<N; i++) {
    //     check_cuda_error(hipFree(*(str_arr+i)));
    //     check_cuda_error(hipFree(*(sorted_arr+i)));
    // }
    hipFree(d_str_arr);
    hipFree(d_sorted_arr);
    hipFree(d_result);
    hipFree(d_N);

    return result;
}


void bubble_sort(int N, char **str_arr) {
    char temp_str[30];

    for(int i=1; i<N; i++)
    {
        for(int j=1; j<N; j++)
        {
            if(strncmp(str_arr[j-1], str_arr[j],30)>0)
            {
                strncpy(temp_str, str_arr[j-1], 30);
                strncpy(str_arr[j-1], str_arr[j], 30);
                strncpy(str_arr[j], temp_str, 30);
            }
        }
    }
}


__global__ void __gpu_radix_sort_init_arr_kernel(int *count, int *offset, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        offset[tid] = 0;
    if (tid < MAX_CHAR-MIN_CHAR+1)
        count[tid] = 0;
}

__global__ void __gpu_radix_sort_count_kernel(char **str_arr, int *count, int *offset, int N, int pos) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        int index = (pos < __strlen_kernel(str_arr[tid])) ? __char_to_index_kernel(str_arr[tid][pos]) : 0;
        offset[tid] = count[index];
        atomicAdd(&count[index], 1);
    }
}

__global__ void __gpu_radix_sort_prefix_sum_kernel(int *count, int N) {
    for (int i=1; i<MAX_CHAR-MIN_CHAR+1; i++) {
        count[i] += count[i-1];
    }
}

__global__ void __gpu_radix_sort_reorder_kernel(char **str_arr, char **output_arr, int *count, int *index, int N, int pos) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        int str_len = __strlen_kernel(str_arr[tid]);
        int c_index = (pos < str_len) ? __char_to_index_kernel(str_arr[tid][pos]) : 0;
        int o_index = count[c_index] + index[tid];
        output_arr[o_index] = str_arr[tid];
    }
}

__global__ void __gpu_radix_sort_copy_arr_kernel(char **str_arr, char **output_arr, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        str_arr[tid] = output_arr[tid];
    }
}

void gpu_radix_sort(int block_size, int N, char **str_arr) {
    int *d_count, *d_offset, *d_N, *d_i;
    char **d_str_arr, **d_output;
    char *d_str;

    print_mem();

    printf("check\n");
    check_cuda_error(hipMalloc((void**)&d_count, (MAX_CHAR-MIN_CHAR+1) * sizeof(int)));
    printf("check\n");
    check_cuda_error(hipMalloc((void**)&d_str_arr, N * sizeof(char *)));
    printf("check\n");
    check_cuda_error(hipMalloc((void**)&d_output, N * sizeof(char *)));
    printf("check\n");
    check_cuda_error(hipMalloc((void**)&d_offset, N * sizeof(int)));
    printf("check\n");
    check_cuda_error(hipMalloc((void**)&d_N, sizeof(int)));
    printf("check\n");
    check_cuda_error(hipMalloc((void**)&d_i, sizeof(int)));
    printf("check\n");

    check_cuda_error(hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice));
    printf("check\n");

    for (int i=0; i<N; i++) {
        check_cuda_error(hipMalloc((void**)&d_str, MAX_STR_LEN * sizeof(char)));
        check_cuda_error(hipMemcpy(d_str, str_arr[i], MAX_STR_LEN * sizeof(char), hipMemcpyHostToDevice));
        check_cuda_error(hipMemcpy(d_str_arr+i, &d_str, sizeof(char *), hipMemcpyHostToDevice));
    }

    int block_num = (N + block_size - 1) / block_size;

    printf("start sorting\n");
    for (int i=0; i<MAX_STR_LEN; i++) {
        check_cuda_error(hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice));
        // printf("i: %d\t", i);
        __gpu_radix_sort_init_arr_kernel<<<block_num, block_size>>>(d_count, d_offset, *d_N);
        printf("init done\t");

        __gpu_radix_sort_count_kernel<<<1, 1>>>(d_str_arr, d_count, d_offset, *d_N, *d_i);
        check_cuda_error(hipDeviceSynchronize());
        printf("count, offset done\t");
        
        __gpu_radix_sort_prefix_sum_kernel<<<1, 1>>>(d_count, *d_N);
        check_cuda_error(hipDeviceSynchronize());
        printf("prefix sum done\t");

        __gpu_radix_sort_reorder_kernel<<<block_num, block_size>>>(d_str_arr, d_output, d_count, d_count, *d_N, *d_i);
        check_cuda_error(hipDeviceSynchronize());
        printf("reorder done\n");

        __gpu_radix_sort_copy_arr_kernel<<<block_num, block_size>>>(d_output, d_str_arr, *d_N);
        printf("copy done\n");
    }

    for (int i=0; i<N; i++) {
        check_cuda_error(hipMemcpy(str_arr[i], d_str_arr+i, MAX_STR_LEN * sizeof(char), hipMemcpyDeviceToHost));
        // check_cuda_error(hipFree(*(str_arr+i)));
    }

    hipFree(d_count);
    hipFree(d_str_arr);
    hipFree(d_output);
    hipFree(d_offset);
}